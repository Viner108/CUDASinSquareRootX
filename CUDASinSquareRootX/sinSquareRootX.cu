#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h> 
#include <math.h> 

#define N (1024*1024)

// �������� ���� ����������� �� GPU � ������ ������� �� CPU
__global__ void sinSquareRootX(float* dA)
{
    // ���������� ����� ���� � ����, �� ���� ����� ������� �������� �������
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // ���������� ������� ��� ������� ��������
    float x = 2.0f * 3.1415926f * (float)idx / (float)N;
    dA[idx] = sinf(sqrtf(x));
    // ������ ����� ��������� ���������� �������� � ���� ������ ������
}
 
// ������� ����������� �� host 
int main(int argc, char* argv[])
{
    //��������� ��� ���������� ���������� h = Host
    float* hA;
    //��������� ��� ������� ������ �� ���������� d = Device
    float* dA;
    // ��������� ������ �� host
    hA = (float*)malloc(N * sizeof(float));
    // ��������� ������ �� device
    hipMalloc((void**)&dA, N * sizeof(float)); 
    // ����� ������� ���� GPU � �������� ����������� ������ � ����������� ����� � �����
    sinSquareRootX <<< N / 512, 512 >>> (dA);
    // ������������ ������ �� GPU �� CPU
    hipMemcpy(hA, dA, N * sizeof(float), hipMemcpyDeviceToHost);
    //���������� �������� � ���������
    for (int idx = 0; idx < N; idx++) {
        printf("a[%d] = %.5f\n", idx, hA[idx]);
    }
    // ����������� ������ �� host � device
    free(hA);
    hipFree(dA);
    return 0;
}

