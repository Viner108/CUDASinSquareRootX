#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.h" // �������� ������������ ���� � ����������� �������

#define N (1024*1024)

// ������� ����������� �� host 
int main(int argc, char* argv[])
{
    //��������� ��� ���������� ���������� h = Host
    float* hA;

    //��������� ��� ������� ������ �� ���������� d = Device
    float* dA;

    // ��������� ������ �� host
    hA = (float*)malloc(N * sizeof(float));

    // ��������� ������ �� device � �������� �� ������
    hipError_t err;
    err = hipMalloc((void**)&dA, N * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Cannot allocate GPU memory: &s\n", hipGetErrorString(err));
        return 1;
    }

    // ����� ������� ���� GPU � �������� ����������� ������ � ����������� ����� � �����
    sinSquareRootX <<< N / 512, 512 >>> (dA);
    //hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
    fprintf(stderr, "Cannot launch CUDA kernel: %s\n", hipGetErrorString(err));
    return 1;
    }


    // ������������ ������ �� GPU �� CPU � �������� �� ������
    err = hipMemcpy(hA, dA, N * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Cannot copy data device/host : &s\n", hipGetErrorString(err));
        return 1;
    }

    //���������� �������� � ���������
    for (int idx = 0; idx < N; idx++) {
        printf("a[%d] = %.5f\n", idx, hA[idx]);
    }

    // ����������� ������ �� host � device
    free(hA);
    hipFree(dA);

    return 0;
}
