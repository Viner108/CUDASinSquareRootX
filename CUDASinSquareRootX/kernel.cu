#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#define N (1024*1024)

__global__ void sinSquareRootX(float* dA)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    float x = 2.0f * 3.1415926f * (float)idx / (float)N;
    dA[idx] = sinf(sqrtf(x));
}
